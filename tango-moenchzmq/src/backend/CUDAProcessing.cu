#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_cooperative_groups.h>

#include <iostream>
#include <memory>
#include <string>
#include <random>

constexpr int cluster_size = 3;
constexpr int block_size = 22;
constexpr int grid_size = 20;
constexpr int nsigma = 1;
constexpr int c2 = (cluster_size + 1) / 2;
constexpr int c3 = cluster_size;
constexpr int BUFFER_SIZE = 100;
constexpr int IMAGE_HEIGHT = 400;
constexpr int IMAGE_WIDTH = 400;
constexpr int IMAGE_SIZE = IMAGE_HEIGHT * IMAGE_WIDTH;

__global__ void process_group2D(int* raw_frame, int* class_output_array, float* pedestal_sum, float* pedestal_squared_sum, int* pedestal_counter, bool isPedestal, int *blockIds, int* threadIds){
    
    int pixel_rms = 0;
    // for the most left and most right block we need take only one extra pixel from either right or left
    // for the blocks inbetween we need to take both left and right extra overlapping pixels  
    __shared__ int no_bkgd_for_block[block_size * block_size];
    // depending on the block position they can be unmodified
    // so need set them explictly for 0 !!! ONCE IN BLOCK !!!
    int local_x = threadIdx.x; // in [0, 1, 2, ... , 20, 21]
    int local_y = threadIdx.y; // in [0, 1, 2, ... , 20, 21]
    int global_x = (threadIdx.x + gridDim.x * blockIdx.x) - 1; // 20 * [0, 1, 2, 3, ... , 19] -> global x maps 0 ... 399
    int global_y = (threadIdx.y + gridDim.y * blockIdx.y) - 1; // 20 * [0, 1, 2, 3, ... , 19] -> global y maps 0 ... 399
    // top left column is (threadIdx.x == 0 && blockIdx.x == 0)
    // top row is (threadIdx.y == 0 && blockIdx.y == 0)
    // top right column is (threadIdx.x == (block_size - 1) && blockIdx.x == (grid_size - 1))
    // bottom row is (threadIdx.y == (block_size - 1) && blockIdx.y == (grid_size - 1))
    if (blockIdx.x == 0 && threadIdx.x == 0   ||
        (threadIdx.y == 0 && blockIdx.y == 0) ||
        (threadIdx.x == (block_size - 1) && blockIdx.x == (grid_size - 1)) ||
        (threadIdx.y == (block_size - 1) && blockIdx.y == (grid_size - 1))
       ){
        no_bkgd_for_block[local_y * block_size + local_x] = 0;
    } else {
        // avoid 0 division
        int pixel_counter = max(pedestal_counter[global_y * 400 + global_x], 1);
        int pixel_offset = pedestal_sum[global_y * 400 + global_x] / pixel_counter;
        pixel_rms = sqrt(pedestal_squared_sum[global_y * 400 + global_x] / pixel_counter - pow(pixel_offset, 2));
        no_bkgd_for_block[local_y * block_size + local_x] = raw_frame[global_y * 400 + global_x] - pixel_offset;
    }
    __syncthreads();

    // zones for overlapping (border of the blocks) 
    if (threadIdx.x == 0 ||
        threadIdx.x == (block_size - 1) ||
        threadIdx.y == 0 ||
        threadIdx.y == (block_size - 1)) return;

    // we still need to update pedestal in the global memory
    // actually we calculated each overlapping pixel twice but
    // need to update it only once
    // so we update it from the block where the given pixel is the most right => where this corresponds to 21th pixel

    int v, tot, tl, tr, bl, br, max_value = 0;
    int pixel_class = 0;
    // considering the global_x we are now inspecting the if a pixel global_x a photon_max -> etc
    // 0 - pedestal, 1- photon,  2 - photon_max, 3 - negative_pedestal
    if (!isPedestal){
        for (int dy = - cluster_size / 2; dy < cluster_size / 2 + 1; dy++){
            for (int dx = - cluster_size / 2; dx < cluster_size / 2 + 1; dx++){
                v = no_bkgd_for_block[(local_y + dy) * block_size + (local_x + dx)];
                tot += v;
                if (dx <= 0 && dy <=0) tl+= v;
                if (dx <= 0 && dy >=0) bl+= v;
                if (dx >= 0 && dy <=0) tr+= v;
                if (dx >= 0 && dy >=0) br+= v;
                if (v > max_value) max_value = v;
            }
        }
        // considering negative pedestal 
        if (no_bkgd_for_block[local_y * block_size + local_x] < -nsigma * pixel_rms) {
                class_output_array[global_y * 400 + global_x] = 3;
            // considering candidates which are some kind of photons (either partial photon charges or the center ) 
            } else if (max_value > nsigma * pixel_rms ||
                       // max value of clusters
                       max(tl, max(bl, max(tr, br))) > c2 * nsigma * pixel_rms ||
                       tot > c3 * nsigma * pixel_rms){
                class_output_array[global_y * 400 + global_x] = 3;
                // if the pixel candidate is a max_value in the area
                if (no_bkgd_for_block[local_y * block_size + local_x] == max_value){
                    class_output_array[global_y * 400 + global_x] = 2;
                }
            }
    }
    // since we updating only pixels on the right side we ignore the 0th pixel and 21th pixel in the last block
    // if the pixel_class remained unchanged and not associated with any kind of photons
    // it is a pedestal and need to be added to stats
    if (pixel_class == 0) {
        blockIds[global_x] = blockIdx.x;
        threadIds[global_x] = threadIdx.x;
        int pixel_counter = pedestal_counter[global_y * 400 + global_x]++;
        if (pixel_counter < BUFFER_SIZE){
            pedestal_sum[global_y * 400 + global_x] += raw_frame[global_y * 400 + global_x];
            pedestal_squared_sum[global_y * 400 + global_x] += pow(raw_frame[global_y * 400 + global_x], 2);
        }
        else {
            pedestal_sum[global_y * 400 + global_x] += raw_frame[global_y * 400 + global_x] - pedestal_sum[global_y * 400 + global_x] / BUFFER_SIZE;
            pedestal_squared_sum[global_y * 400 + global_x] += pow(raw_frame[global_y * 400 + global_x], 2) - pedestal_squared_sum[global_y * 400 + global_x] / BUFFER_SIZE;
        }
    }
}

int main(){
    int* test_array_h = new int[IMAGE_SIZE];
    int* output_array_h = new int[IMAGE_SIZE];
    int* input_array_d;
    int* output_array_d;
    float* pedestal_sum_h = new float[IMAGE_SIZE];
    float* pedestal_squared_h = new float[IMAGE_SIZE];
    int* pedestal_counter_h = new int[IMAGE_SIZE];
    int* threadIds_h = new int[IMAGE_SIZE];
    int* blockIds_h = new int[IMAGE_SIZE];

    float* pedestal_sum_d;
    float* pedestal_squared_d;
    int* pedestal_counter_d;
    int* threadIds_d;
    int* blockIds_d;
    hipMalloc(&input_array_d, IMAGE_SIZE*sizeof(int));
    hipMalloc(&output_array_d, IMAGE_SIZE*sizeof(int));
    hipMalloc(&pedestal_sum_d, IMAGE_SIZE*sizeof(float));
    hipMalloc(&pedestal_squared_d, IMAGE_SIZE*sizeof(float));
    hipMalloc(&pedestal_counter_d, IMAGE_SIZE*sizeof(int));
    hipMalloc(&threadIds_d, IMAGE_SIZE*sizeof(int));
    hipMalloc(&blockIds_d, IMAGE_SIZE*sizeof(int));

    hipMemcpy(input_array_d, test_array_h, IMAGE_SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(output_array_d, output_array_h, IMAGE_SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(pedestal_sum_d, 0, IMAGE_SIZE*sizeof(float));
    hipMemset(pedestal_squared_d, 0, IMAGE_SIZE*sizeof(float));
    hipMemset(pedestal_counter_d, 0, IMAGE_SIZE*sizeof(int));
    hipMemset(threadIds_d, 0, IMAGE_SIZE*sizeof(int));
    hipMemset(blockIds_d, 0, IMAGE_SIZE*sizeof(int));
    std::random_device rd{};
    std::mt19937 gen{rd()};
    std::normal_distribution d{100., 10.0};
    // seems like threads per block need to be + (cluster_size / 2) to blocks per grid
    for (int i = 0; i<200; i++){
        for (int y = 0; y < IMAGE_HEIGHT; y++){
            for (int x = 0; x < IMAGE_WIDTH; x++){
                int value = std::round(d(gen));
                test_array_h[y * IMAGE_HEIGHT + x] = value;
            }
        }
        hipMemcpy(input_array_d, test_array_h, IMAGE_SIZE*sizeof(int), hipMemcpyHostToDevice);
        // 22 threads per block is block size + (cluster_size - 1)
        dim3 threadsPerBlock(22, 22);
        dim3 blocksPerGrid(20, 20);
        process_group2D<<<blocksPerGrid, threadsPerBlock>>>(input_array_d, output_array_d, pedestal_sum_d, pedestal_squared_d, pedestal_counter_d, true, blockIds_d, threadIds_d);
    }
    hipMemcpy(pedestal_sum_h, pedestal_sum_d, IMAGE_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(pedestal_squared_h, pedestal_squared_d, IMAGE_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(pedestal_counter_h, pedestal_counter_d, IMAGE_SIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(blockIds_h, blockIds_d, IMAGE_SIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(threadIds_h, threadIds_d, IMAGE_SIZE*sizeof(int), hipMemcpyDeviceToHost);

    for (int x = 0; x<41; x++){
        printf("pedestal_sum %f, pedestal_sum_sqaured %f, counter %d, blockId %d, threadId %d\n", pedestal_sum_h[x], pedestal_squared_h[x], pedestal_counter_h[x], blockIds_h[x], threadIds_h[x]);
    }
    return 0;
}
